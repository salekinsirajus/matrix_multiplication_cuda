/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// A contains input
__global__ void matrix_mul(float *A, float *C, float a, float b, float c, float d, int state_size, int t_bit){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //i=x1, flipped=x2
    int flipped = ((1 << t_bit) | i);
    if (i < state_size){
        if (flipped  > i){
            C[i] = (A[i] * a ) + (A[flipped] * b);
            C[flipped] = (A[i] * c ) + (A[flipped] * d);
        }
    }
   
}


/**
 * Host main routine
 */
int
main(int argc, char **argv)
{
    /*
        INPUT READING FROM FILE
    */
     float a, b, c, d; // quantum gate matrix 
     int t_bit; 
     int vector_size=pow(2,30), final_size;
     float*  state_vector  = (float*)malloc(vector_size * sizeof(float));

     float number;
      FILE* in_file = fopen(argv[1], "r"); // read only  
     
      if (! in_file ) // equivalent to saying if ( in_file == NULL ) 
         {  
            printf("oops, file can't be read\n"); 
            exit(-1); 
         } 

      // attempt to read the next line and store 
      // the value in the "number" variable 
      int count = 0;
      float last;
      while ( fscanf(in_file, "%f", &number ) != EOF )  
         { 
           if (count==0) a = number;
           else if (count==1) b = number;
           else if (count==2) c = number;
           else if (count==3) d = number;

           else {
               state_vector[count-4] = number; 
            }

           last = number;
           count++;
         } 
           
     t_bit = (int) last;
     final_size = count - 5;
    //printf("Qubit %d size: %d a:%f, b:%f c:%f d:%f\n", qubit, final_size, a, b, c, d);
    /* END READING FROM INPUT */


    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = final_size;
    size_t size = numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = state_vector[i]; 
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMallocManaged((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMallocManaged((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    //printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    matrix_mul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, a, b, c, d, numElements, t_bit);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        printf("%.3f\n", h_C[i]);
    }


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_C);

    //Free IO memory
    free(state_vector);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}


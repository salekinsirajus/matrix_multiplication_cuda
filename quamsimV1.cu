/**
 * File: quamsimV1.cu
 * Author: Sirajus Salekin
 *
 * Contains cuda kernel that performs matrix multiplication to simulate
 * applying quantum gate on a quantum circuit.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// A contains input
__global__ void matrix_mul(float *A, float *C, float a, float b, float c, float d, int state_size, int t_bit){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //i=x1, flipped=x2
    int flipped = ((1 << t_bit) | i);
    if (i < state_size){
        if (flipped  > i){
            C[i] = (A[i] * a ) + (A[flipped] * b);
            C[flipped] = (A[i] * c ) + (A[flipped] * d);
        }
    }
}


/**
 * Host main routine
 */
int main(int argc, char **argv){
    /*
        INPUT READING FROM FILE
    */
     float a, b, c, d; // quantum gate matrix
     int t_bit;
     int vector_size=pow(2,30), final_size;
     float* state_vector  = (float*)malloc(vector_size * sizeof(float));

     float number;
     FILE* in_file = fopen(argv[1], "r"); // read only

     if (!in_file ){// equivalent to saying if ( in_file == NULL )
        printf("oops, file can't be read\n");
        exit(-1);
     }

     // attempt to read the next line and store
     // the value in the "number" variable
     int count = 0;
     float last;
     while ( fscanf(in_file, "%f", &number ) != EOF ){
          if (count==0) a = number;
          else if (count==1) b = number;
          else if (count==2) c = number;
          else if (count==3) d = number;

          else {
              state_vector[count-4] = number;

           }

          last = number;
          count++;
        }

     t_bit = (int) last;
     final_size = count - 5;
     //printf("Qubit %d size: %d a:%f, b:%f c:%f d:%f\n", qubit, final_size, a, b, c, d);
     /* END READING FROM INPUT */


     // Error code to check return values for CUDA calls
     hipError_t err = hipSuccess;

     // Print the vector length to be used, and compute its size
     int numElements = final_size;
     size_t size = numElements * sizeof(float);

     // Allocate the host input vector A
     float *h_A = (float *)malloc(size);

     // Allocate the host output vector C
     float *h_C = (float *)malloc(size);

     // Verify that allocations succeeded
     if (h_A == NULL || h_C == NULL){
         fprintf(stderr, "Failed to allocate host vectors!\n");
         exit(EXIT_FAILURE);
     }

     // Initialize the host input vectors
     for (int i = 0; i < numElements; ++i){
         h_A[i] = state_vector[i];
     }

     // Allocate the device input vector A
     float *d_A = NULL;
     err = hipMalloc((void **)&d_A, size);

     if (err != hipSuccess){
         fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }


     // Allocate the device output vector C
     float *d_C = NULL;
     err = hipMalloc((void **)&d_C, size);

     if (err != hipSuccess){
         fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // Copy the host input vectors A and B in host memory to the device input vectors in
     // device memory
     err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

     if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }


     // Launch the CUDA Kernel
     int threadsPerBlock = 256;
     int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
     //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
     matrix_mul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, a, b, c, d, numElements, t_bit);
     err = hipGetLastError();

     if (err != hipSuccess){
         fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // Copy the device result vector in device memory to the host result vector
     // in host memory.
     err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
     if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // print out the results stored in h_C
     for (int i = 0; i < numElements; ++i){
         printf("%.3f\n", h_C[i]);
     }


     // Free device global memory
     err = hipFree(d_A);
     if (err != hipSuccess){
         fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }


     err = hipFree(d_C);
     if (err != hipSuccess){
         fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // Free host memory
     free(h_A);
     free(h_C);

     //Free IO memory
     free(state_vector);

     // Reset the device and exit
     // cudaDeviceReset causes the driver to clean up all state. While
     // not mandatory in normal operation, it is good practice.  It is also
     // needed to ensure correct operation when the application is being
     // profiled. Calling cudaDeviceReset causes all profile data to be
     // flushed before the application exits
     err = hipDeviceReset();

     if (err != hipSuccess){
         fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     return 0;
}


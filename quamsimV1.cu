/**
 * File: quamsimV1.cu
 * Author: Sirajus Salekin
 *
 * Contains cuda kernel that performs matrix multiplication to simulate
 * applying quantum gate on an n-qubit quantum circuit.
 *
 * https://dl.acm.org/doi/abs/10.1145/3447818.3460357
 *  
 * See section 2 in this paper for details. Motivation behind the algorithm
 * implemented here is directly quoted from the paper:
 *
 * "An n-qubit system is represented by 2^n complex numbers. A gate on
 * the n-qubit system is a 2^n x 2^n unitary matrix. However, the 2^n x 2^n 
 * matrix representations of single qubit gates and controlled gates are
 * very sparse and there is no need to construct the matrix. Instead,
 * these gates can be implemented by several small matrix multiplications.
 * ...
 * And applying a two-qubit controlled gate on the n-qubit system
 * is equivalent to 2^(n−2) matrix multiplication tasks. Each task updates
 * two positions with the index of the control qubit C equals 1 and
 * only the target qubit t differs."
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 *
 * The kernel code takes the thread id and use it to calculate the pair of elements
 * in the quantum state that should be computed together. It checks for 1 or 0 at the "t-th" bit
 * and calculates the matrix once it finds that. However, when it figures out that pairing
 * criteria does not match, that thread does not do anything. Some optimization that can be done here
 *
 * Inputs:
 *  1. float *A: statevector of the n-qubit system
 *  2. float *C: statevector after applying the quantum gate operation 
 *  3. float a, b, c, d: represent the quantum gate being applied to
 *     |a b|
 *     |c d|
 *  4. state_size: size of the statevector pointed by *A and *C
 *  5. t-bit: The bit where the target qubit differs.
 */

__global__ void matrix_mul(float *A, float *C, float a, float b, float c, float d, int state_size, int t_bit){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //i=x1, flipped=x2
    int flipped = ((1 << t_bit) | i);
    if (i < state_size){
        if (flipped  > i){
            C[i] = (A[i] * a ) + (A[flipped] * b);
            C[flipped] = (A[i] * c ) + (A[flipped] * d);
        }
    }
}


/**
 * Host main routine
 */
int main(int argc, char **argv){
    /*
        INPUT READING FROM FILE
    */
     float a, b, c, d; // quantum gate matrix
     int t_bit;
     int vector_size=pow(2,30), final_size;
     float* state_vector  = (float*)malloc(vector_size * sizeof(float));

     float number;
     FILE* in_file = fopen(argv[1], "r"); // read only

     if (!in_file ){// equivalent to saying if ( in_file == NULL )
        printf("oops, file can't be read\n");
        exit(-1);
     }

     // attempt to read the next line and store
     // the value in the "number" variable
     int count = 0;
     float last;
     while ( fscanf(in_file, "%f", &number ) != EOF ){
          if (count==0) a = number;
          else if (count==1) b = number;
          else if (count==2) c = number;
          else if (count==3) d = number;

          else {
              state_vector[count-4] = number;

           }

          last = number;
          count++;
        }

     t_bit = (int) last;
     final_size = count - 5;
     //printf("Qubit %d size: %d a:%f, b:%f c:%f d:%f\n", qubit, final_size, a, b, c, d);
     /* END READING FROM INPUT */


     // Error code to check return values for CUDA calls
     hipError_t err = hipSuccess;

     // Print the vector length to be used, and compute its size
     int numElements = final_size;
     size_t size = numElements * sizeof(float);

     // Allocate the host input vector A
     float *h_A = (float *)malloc(size);

     // Allocate the host output vector C
     float *h_C = (float *)malloc(size);

     // Verify that allocations succeeded
     if (h_A == NULL || h_C == NULL){
         fprintf(stderr, "Failed to allocate host vectors!\n");
         exit(EXIT_FAILURE);
     }

     // Initialize the host input vectors
     for (int i = 0; i < numElements; ++i){
         h_A[i] = state_vector[i];
     }

     // Allocate the device input vector A
     float *d_A = NULL;
     err = hipMalloc((void **)&d_A, size);

     if (err != hipSuccess){
         fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }


     // Allocate the device output vector C
     float *d_C = NULL;
     err = hipMalloc((void **)&d_C, size);

     if (err != hipSuccess){
         fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // Copy the host input vectors A and B in host memory to the device input vectors in
     // device memory
     err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

     if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }


     // Launch the CUDA Kernel
     int threadsPerBlock = 256;
     int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
     //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
     matrix_mul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, a, b, c, d, numElements, t_bit);
     err = hipGetLastError();

     if (err != hipSuccess){
         fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // Copy the device result vector in device memory to the host result vector
     // in host memory.
     err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
     if (err != hipSuccess){
         fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // print out the results stored in h_C
     for (int i = 0; i < numElements; ++i){
         printf("%.3f\n", h_C[i]);
     }


     // Free device global memory
     err = hipFree(d_A);
     if (err != hipSuccess){
         fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }


     err = hipFree(d_C);
     if (err != hipSuccess){
         fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     // Free host memory
     free(h_A);
     free(h_C);

     //Free IO memory
     free(state_vector);

     // Reset the device and exit
     // cudaDeviceReset causes the driver to clean up all state. While
     // not mandatory in normal operation, it is good practice.  It is also
     // needed to ensure correct operation when the application is being
     // profiled. Calling cudaDeviceReset causes all profile data to be
     // flushed before the application exits
     err = hipDeviceReset();

     if (err != hipSuccess){
         fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
         exit(EXIT_FAILURE);
     }

     return 0;
}

